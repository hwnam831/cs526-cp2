/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <math.h>
#include <hip/hip_runtime.h>
#include <builtin_types.h>
#include "nvvm.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/stat.h>
#include <string.h>
#include <iostream>
#include <fstream>
#include <hipblas.h>
#include "strsm_gold.cpp"

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

// These are the inline versions for all of the SDK helper functions
void __checkCudaErrors( hipError_t err, const char *file, const int line )
{
    if( hipSuccess != err) {
        fprintf(stderr, "%s(%i) : cudaSafeCall() Runtime API error %d: %s.\n",
                file, line, (int)err, hipGetErrorString( err ) );
        exit(-1);
    }
}

void __checkCudaErrors( hipError_t err, const char *file, const int line )
{
    if( hipSuccess != err) {
        fprintf(stderr, "checkCudaErrors() Driver API error = %04d \from file <%s>, line %i.\n",
                err, file, line );

        const char *p;
        hipDrvGetErrorString(err, &p);
        fprintf(stderr, "%s\n", p);
        exit(-1);
    }
}

hipDevice_t cudaDeviceInit()
{
    hipDevice_t cuDevice = 0;
    int deviceCount = 0;
    hipError_t err = hipInit(0);
    char name[100];
    int major=0, minor=0;

    if (hipSuccess == err)
        checkCudaErrors(hipGetDeviceCount(&deviceCount));
    if (deviceCount == 0) {
        fprintf(stderr, "cudaDeviceInit error: no devices supporting CUDA\n");
        exit(-1);
    }
    checkCudaErrors(hipDeviceGet(&cuDevice, 0));
    hipDeviceGetName(name, 100, cuDevice);
    printf("Using CUDA Device [0]: %s\n", name);

    checkCudaErrors( hipDeviceComputeCapability(&major, &minor, cuDevice) );
    if (major < 2) {
        fprintf(stderr, "Device 0 is not sm_20 or later\n");
        exit(-1);
    }
    return cuDevice;
}


hipError_t initCUDA(hipCtx_t *phContext,
                  hipDevice_t *phDevice,
                  hipModule_t *phModule,
                  hipFunction_t *phKernel,
                  const char *ptx,
                  const char *kernelname)
{
    // Initialize 
    *phDevice = cudaDeviceInit();

    // Create context on the device
    checkCudaErrors(hipCtxCreate(phContext, 0, *phDevice));

    // Load the PTX 
    checkCudaErrors(hipModuleLoadDataEx(phModule, ptx, 0, 0, 0));

    // Locate the kernel entry poin
    checkCudaErrors(hipModuleGetFunction(phKernel, *phModule, kernelname));


    return hipSuccess;
}

char *loadProgramSource(const char *filename, size_t *size) 
{
    struct stat statbuf;
    FILE *fh;
    char *source = NULL;
    *size = 0;
    fh = fopen(filename, "rb");
    if (fh) {
        stat(filename, &statbuf);
        source = (char *) malloc(statbuf.st_size+1);
        if (source) {
            fread(source, statbuf.st_size, 1, fh);
            source[statbuf.st_size] = 0;
            *size = statbuf.st_size+1;
        }
    }
    else {
        fprintf(stderr, "Error reading file %s\n", filename);
        exit(-1);
    }
    return source;
}

char *generatePTX(const char *ll, size_t size, const char *filename)
{
    nvvmResult result;
    nvvmProgram program;
    size_t PTXSize;
    char *PTX = NULL;

    result = nvvmCreateProgram(&program);
    if (result != NVVM_SUCCESS) {
        fprintf(stderr, "nvvmCreateProgram: Failed\n");
        exit(-1); 
    }

    result = nvvmAddModuleToProgram(program, ll, size, filename);
    if (result != NVVM_SUCCESS) {
        fprintf(stderr, "nvvmAddModuleToProgram: Failed\n");
        exit(-1);
    }
 
    result = nvvmCompileProgram(program,  0, NULL);
    if (result != NVVM_SUCCESS) {
        char *Msg = NULL;
        size_t LogSize;
        fprintf(stderr, "nvvmCompileProgram: Failed\n");
        nvvmGetProgramLogSize(program, &LogSize);
        Msg = (char*)malloc(LogSize);
        nvvmGetProgramLog(program, Msg);
        fprintf(stderr, "%s\n", Msg);
        free(Msg);
        exit(-1);
    }
    
    result = nvvmGetCompiledResultSize(program, &PTXSize);
    if (result != NVVM_SUCCESS) {
        fprintf(stderr, "nvvmGetCompiledResultSize: Failed\n");
        exit(-1);
    }
    
    PTX = (char*)malloc(PTXSize);
    result = nvvmGetCompiledResult(program, PTX);
    if (result != NVVM_SUCCESS) {
        fprintf(stderr, "nvvmGetCompiledResult: Failed\n");
        free(PTX);
        exit(-1);
    }
    
    result = nvvmDestroyProgram(&program);
    if (result != NVVM_SUCCESS) {
      fprintf(stderr, "nvvmDestroyProgram: Failed\n");
      free(PTX);
      exit(-1);
    }
    
    return PTX;
}

// Allocates a matrix with random float entries.
void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

int checkarray(float* reference, float* o_data, int num_elements) {
    {
        int error = 0;
        for (int i=0; i<num_elements; i++) {
            for (int j=0; j<num_elements; j++) {
                float t = reference[j*num_elements+i]-o_data[j*num_elements+i];
                if (t<0) t = -t;
                float ref = reference[j*num_elements+i];
                if  (ref<0) ref = -ref;
                if (t/ref>1e-3) {
                    if (error<4)
                        printf("%d, %d, %f, %f\n", i, j, reference[j*num_elements+i], o_data[j*num_elements+i]);
                    error++;
                }
            }
        }
        return error;
    }
}

void printDiff(float *data1, float *data2, int width, int height)
{
  int i,j,k;
  int error_count=0;
  for (j=0; j<height; j++) {
    for (i=0; i<width; i++) {
      k = j*width+i;
      if (data1[k] != data2[k]) {
         printf("diff(%d,%d) CPU=%4.4f, GPU=%4.4f n", i,j, data1[k], data2[k]);
         error_count++;
      }
    }
  }
  printf(" nTotal Errors = %d n", error_count);
}

#define INPUT_WIDTH 8192
int main(int argc, char **argv)
{
    const unsigned int nThreads = 32;
    const unsigned int nBlocks  = 1;
    const size_t memSize = nThreads * nBlocks * sizeof(int);

    hipCtx_t    hContext = 0;
    hipDevice_t     hDevice  = 0;
    hipModule_t     hModule  = 0;
    hipFunction_t   hKernel  = 0;
    hipDeviceptr_t  d_A   = 0;
    hipDeviceptr_t  d_B   = 0;
    hipDeviceptr_t  d_C   = 0;
    float*   h_A   = 0;
    float*   h_B   = 0;
    float*   h_C   = 0;
    char        *ptx      = NULL;
    unsigned int i;

    // Get the ll from file
    size_t size = 0;
    // Kernel parameters
    if (argc < 3){
        fprintf(stdout, "Usage: ./loader [PTXFILE] [KERNELNAME]");
        return -1;
    }
    
    const char *filename = argv[1];

    char *ll = loadProgramSource(filename, &size);
    fprintf(stdout, "NVVM IR ll file loaded\n");

    // Use libnvvm to generte PTX
    ptx = loadProgramSource(filename, &size);
    fprintf(stdout, "PTX generated:\n");
    fprintf(stdout, "%s\n", ptx);
/*
    std::ifstream t(filename);
    if(!t.is_open()) {
        fprintf(stderr, "file not found\n");
        exit(-1);
    }
    std::string str((std::istreambuf_iterator<char>(t)), std::istreambuf_iterator<char>());
    fprintf(stdout, "%s\n", str.c_str());
*/
    // Initialize the device and get a handle to the kernel
    checkCudaErrors(initCUDA(&hContext, &hDevice, &hModule, &hKernel, ptx, argv[2] ));

    unsigned int num_elements = INPUT_WIDTH;

    // allocate host memory for matrices A and B
    const unsigned int in_mem_size = sizeof( float) * (num_elements*num_elements);
    const unsigned int out_mem_size = sizeof( float) * (num_elements*num_elements);
    if ((h_A = (float*) malloc(in_mem_size)) == NULL) {
        fprintf(stderr, "Could not allocate host memory\n");
        exit(-1);
    }
    if ((h_B = (float*) malloc(in_mem_size)) == NULL) {
        fprintf(stderr, "Could not allocate host memory\n");
        exit(-1);
    }

    // initialize host memory
    for( unsigned int i = 0; i < num_elements; ++i)
    {
        for( unsigned int j = 0; j < num_elements; ++j) {
            h_A[i*num_elements+j] = ((rand()/(float)RAND_MAX));
            if (i>j) h_A[i*num_elements+j]=0.0f;
            h_B[i*num_elements+j] = ((rand()/(float)RAND_MAX));
        }
    }

    // allocate host memory for the result
    if ((h_C = (float*) malloc(out_mem_size)) == NULL) {
        fprintf(stderr, "Could not allocate host memory\n");
        exit(-1);
    }

    checkCudaErrors(hipMalloc((void**) &d_A, in_mem_size));
    checkCudaErrors(hipMalloc((void**) &d_C, out_mem_size));

    // copy host memory to device
    checkCudaErrors(hipMemcpy((void*) d_A, h_A, in_mem_size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy((void*) d_C, h_B, in_mem_size, hipMemcpyHostToDevice));

    float* reference = (float*) malloc(out_mem_size);
    if (reference == NULL) {
        fprintf(stderr, "Could not allocate reference memory\n");
        exit(-1);
    }

    computeGold(h_A, h_B, num_elements, reference);

    // setup execution parameters
    int block_width = 256;

    hipblasStrsm('L', 'L', 'N', 'N', num_elements, num_elements, 1.0, (float*)d_A, num_elements, (float*)d_C, num_elements);
    hipDeviceSynchronize();
    checkCudaErrors(hipMemcpy(reference, (void*) d_C, out_mem_size, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy((void*) d_A, h_A, in_mem_size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy((void*) d_C, h_B, in_mem_size, hipMemcpyHostToDevice));
    
    for (int i=0; i<num_elements; i+=block_width) {
        hipblasStrsm('L', 'L', 'N', 'N', block_width, num_elements, 1.0, (float*)d_A+i*num_elements+i, num_elements, (float*)d_C+i, num_elements);
        // left matrix (i,i) (i+64, i+64)        right matrix (0,i) (0, i+64)

        // strsm to get the result matrix (0,i) (0, i+64)
        // result(0, i+64) (0, h) - left matrix (i, i+64) (i+64,h) * result matrix (0,i) (0, i+64)
        dim3 threads(block_width, 1);
        int WC = num_elements - i - block_width;
        if (WC==0) break;
        int HC = num_elements;
        dim3 grid(WC / threads.x, HC / threads.y);

        int i_val = i;
        void *params[] = { &d_C, &d_A, &d_C, &block_width, &num_elements, &i_val };
        // Launch the kernel
        checkCudaErrors(hipModuleLaunchKernel(hKernel, grid.x, grid.y, 1, threads.x, threads.y, 1,
                                       0, NULL, params, NULL));
    }

    hipDeviceSynchronize();
    fprintf(stderr, "CUDA kernel launched\n");
    // Copy the result back to the host
    checkCudaErrors(hipMemcpy(h_C, (void*) d_C, out_mem_size, hipMemcpyDeviceToHost));

    // compute reference solution
    
    int res = checkarray(reference, h_C, num_elements);
    printf("Test %s \n", (res == 0) ? "PASSED" : "FAILED");

    if (res != 0) {
        printDiff(reference, h_C,  num_elements, num_elements);
    }
    
    // Cleanup
    checkCudaErrors(hipFree((void *) d_A));
    checkCudaErrors(hipFree((void *) d_B));
    free(h_A);
    free(h_B);
    free(h_C);
    free(reference);
    if (hModule) {
        checkCudaErrors(hipModuleUnload(hModule));
        hModule = 0;
    }
    if (hContext) {
        checkCudaErrors(hipCtxDestroy(hContext));
        hContext = 0;
    }

    //free(ll);
    //free(ptx);
    
    return 0;
}

